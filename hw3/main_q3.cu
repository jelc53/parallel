#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <vector>
#include <stdexcept>
#include <string>
#include <utility>

#include "util.cuh"
#include "benchmark.cuh"

constexpr int MAX_STRIDE = 32;
constexpr int X_FILL = 0;
constexpr int Y_FILL = 1;
constexpr int Z_FILL = -1;

void checkErrors(int *z, unsigned int stride, unsigned int N) {
  for (unsigned int i = 0; i < N; ++i) {
    if (i % stride == 0) {
      if(z[i] != 0x01010101) {
        std::cerr << "Mismatch with stride " << stride << ". " << std::endl;
        std::cerr << "z[" << i << "] != x[" << i <<"] + " << "y[" << i << "]" << std::endl;
        exit(1);
      }
    }
    else {
      if(z[i] != Z_FILL) {
        std::cerr << "Mismatch with stride " << stride << ". " << std::endl;
        std::cerr << "z[" << i << "] != x[" << i <<"] + " << "y[" << i << "]" << std::endl;
        exit(1);
      }
    }
  }
}

void writeToCSV(std::string filename, 
		std::vector<std::pair<std::string, std::vector<double>>> data) {
  // create output filestream object
  std::ofstream myFile(filename);

  // send column names to stream
  for (int j=0; j<data.size(); ++j){
    myFile << data.at(j).first;
    if (j != data.size() -1){
      myFile << ","; // no comma at end of line
    }
  }
  myFile << "\n";

  // send data to stream
  for (int i=0; i<data.at(0).second.size(); ++i) {
    for (int j=0; j<data.size(); ++j) {
      myFile << data.at(j).second.at(i);
      if (j != data.size() -1) {
        myFile << ",";  // no comma at end of line
      }
    }
    myFile << "\n";
  }
  myFile.close();
}

int main(int argc, char **argv) {

  hipDeviceProp_t prop;
  hipError_t err = hipGetDeviceProperties(&prop, 0);
  if (err != hipSuccess)
    throw std::runtime_error("Failed to get CUDA device name");
  std::cout << "# Using device: " << prop.name << std::endl;

  // Set up work vectors
  std::size_t N = 10000000;

  
  int *x, *y, *z;
  int host_z[MAX_STRIDE * 2];

  err = hipMalloc(&x, sizeof(int) * MAX_STRIDE * N);
  if (err != hipSuccess)
    throw std::runtime_error("Failed to allocate CUDA memory for x");
  err = hipMalloc(&y, sizeof(int) * MAX_STRIDE * N);
  if (err != hipSuccess)
    throw std::runtime_error("Failed to allocate CUDA memory for y");
  err = hipMalloc(&z, sizeof(int) * MAX_STRIDE * N);
  if (err != hipSuccess)
    throw std::runtime_error("Failed to allocate CUDA memory for z");

  // Warmup calculation:
  elementwise_add<<<72, 1024>>>(x, y, z, static_cast<unsigned int>(1),
                                static_cast<unsigned int>(N));
  check_launch("warm up");


  std::vector<double> stride_array;
  std::vector<double> time_array;
  std::vector<double> bandwidth_array;
  
  // Benchmark runs
  const int n_repeat = 5;
  printf("# stride     time [ms]   GB/sec\n");
  for (int stride = 1; stride <= MAX_STRIDE; ++stride) {
    event_pair timer;


    start_timer(&timer);
    // repeat calculation several times, then average
    for (int num_runs = 0; num_runs < n_repeat; ++num_runs) {
      elementwise_add<<<72, 1024>>>(x, y, z, static_cast<unsigned int>(stride),
                                    static_cast<unsigned int>(N));  
    }
    double exec_time = stop_timer(&timer);

    check_launch("elementwise_add");


    hipMemset(x, X_FILL, sizeof(int) * MAX_STRIDE * N);
    hipMemset(y, Y_FILL, sizeof(int) * MAX_STRIDE * N);
    hipMemset(z, Z_FILL, sizeof(int) * MAX_STRIDE * N);
    elementwise_add<<<72, 1024>>>(x, y, z, static_cast<unsigned int>(stride),
                                  static_cast<unsigned int>(N));  
    hipDeviceSynchronize();

    printf("   %5d    %8.4f   %7.1f\n", stride, exec_time, n_repeat * 3.0 * sizeof(int) * N / exec_time * 1e-6);
    stride_array.push_back(stride);
    time_array.push_back(exec_time);
    bandwidth_array.push_back(n_repeat*3.0*sizeof(int)*N/exec_time*1e-6);
    
    hipMemcpy(&host_z, z, sizeof(int) * MAX_STRIDE * 2, hipMemcpyDeviceToHost);
    checkErrors(host_z, stride, MAX_STRIDE * 2);

  }


  // write results to csv
  std::vector<std::pair<std::string, std::vector<double>>> csv_out = {
	    {"stride", stride_array},
	    //{"time_ms", time_array}, 
	    {"bandwidth_gbps", bandwidth_array}
  };
  writeToCSV("q3_1.csv", csv_out);
  bandwidth_array.clear();
  time_array.clear();
  stride_array.clear();

  return EXIT_SUCCESS;
}
