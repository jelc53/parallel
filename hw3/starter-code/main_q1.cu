#include "hip/hip_runtime.h"
/* This is machine problem 1, part 1, recurrence problem
 *
 * The problem is to take in the number of iterations and a vector of constants,
 * and perform the recurrence on each constant to determine whether it lies in
 * the (modified) Mandelbrot Set.
 *
 */

#include <math.h>

#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <vector>

#include "util.cuh"
#include "recurrence.cuh"

using std::cerr;
using std::cout;
using std::endl;
using std::fabs;
using std::vector;

typedef float elem_type;
typedef std::vector<elem_type> vec;

constexpr const size_t MAX_ARR_SIZE = (1 << 30);  // NOTE: change this to 100 for debugging

const size_t ITER_MAX_CHECK = 10; 
/* Maximum number of iterations for which error is checked;
   This is to avoid having to consider the accumulation of roundoff errors.
*/

// TODO: initialize an array of size arr_size in input_array with random floats
// between -1 and 1
void initialize_array(vec &input_array, size_t arr_size) {

}

void host_recurrence(vec &input_array, vec &output_array, size_t num_iter,
                     size_t array_size) {
  std::transform(input_array.begin(), input_array.begin() + array_size,
                 output_array.begin(), [&num_iter](elem_type &constant) {
                   elem_type z = 0;
                   for (size_t it = 0; it < num_iter; it++) {
                     z = z * z + constant;
                   }
                   return z;
                 });
}

void check_initialization(vec &input_array, size_t arr_size) {
  if (input_array.size() != arr_size) {
    cerr << "Initialization Error: Array size isn't correct." << endl;
  }

  int count = 0;
  for (size_t i = 0; i < arr_size; i++) {
    elem_type entry = input_array[i];
    if (entry < -1.0 || entry > 1.0) {
      cerr << "Initialization Error: Entry " << i << " isn't between -2 and 2."
           << endl;
      count++;
    }

    if (count > 10) {
      cerr << "Too many (>10) errors in initialization, quitting..." << endl;
      break;
    }
  }
}

void checkResults(vec &array_host, elem_type *device_output_array,
                  size_t num_entries) {
  // allocate space on host for gpu results
  vec array_from_gpu(num_entries);

  // download and inspect the result on the host:
  hipMemcpy(&array_from_gpu[0], device_output_array,
             num_entries * sizeof(elem_type), hipMemcpyDeviceToHost);
  check_launch("copy from gpu");

  // check CUDA output versus reference output
  int error = 0;
  float max_error = 0.;
  int pos = 0;
  double inf = std::numeric_limits<double>::infinity();
  for (size_t i = 0; i < num_entries; i++) {

    double err = fabs(array_host[i]) <= 1 ? 
                    fabs( array_host[i] - array_from_gpu[i] ) :
                    fabs((array_host[i] - array_from_gpu[i]) / array_host[i]);
    if (max_error < err) {
        max_error = err;
        pos = i;      
    }
    if (fabs(array_host[i]) == inf && fabs(array_from_gpu[i]) == inf)
        continue;
    if (fabs(array_host[i]) <= 2 && fabs(array_host[i] - array_from_gpu[i]) < 1e-4)
        continue;
    if (fabs(array_host[i]) > 2 &&
        fabs((array_host[i] - array_from_gpu[i]) / array_host[i]) < 1e-4)
        continue;               

    ++error;
    cerr << "** Critical error at pos: " << i
        << " error "
        << fabs((array_host[i] - array_from_gpu[i]) / array_host[i])    
        << " expected " << array_host[i] << " and got " << array_from_gpu[i]
        << endl;

    if (error > 10) {
      cerr << endl << "Too many critical errors, quitting..." << endl;
      break;
    }
  }

  cout << "Largest error found at pos: " << pos 
    << " error " << max_error
    << " expected " << array_host[pos]
    << " and got "  << array_from_gpu[pos] << endl;

  if (error) {
    cerr << "\nCritical error(s) in recurrence kernel! Exiting..." << endl;
    exit(1);
  }
}

double recurAndCheck(const elem_type *device_input_array,
                     elem_type *device_output_array, size_t num_iter,
                     size_t array_size, size_t cuda_block_size,
                     size_t cuda_grid_size, vec &arr_host) {
  // generate GPU output
  double elapsed_time =
      doGPURecurrence(device_input_array, device_output_array, num_iter,
                      array_size, cuda_block_size, cuda_grid_size);

  if (num_iter <= ITER_MAX_CHECK)
    checkResults(arr_host, device_output_array, array_size);

  // make sure we don't falsely say the next kernel is correct because
  // we've left the correct answer sitting in memory
  hipMemset(device_output_array, 0, array_size * sizeof(elem_type));
  return elapsed_time;
}

int main(int argc, char **argv) {
  int exit_code = 0;

  // init array
  vec init_arr;
  initialize_array(init_arr, MAX_ARR_SIZE);
  check_initialization(init_arr, MAX_ARR_SIZE);

  hipFree(0);  // initialize cuda context to avoid including cost in timings later

  // Warm-up each of the kernels to avoid including overhead in timing.
  // If the kernels are written correctly, then they should
  // never make a bad memory access, even though we are passing in NULL
  // pointers since we are also passing in a size of 0
  recurrence<<<1, 1>>>(nullptr, nullptr, 0, 0);

  // allocate host arrays
  vec arr_gpu(MAX_ARR_SIZE);
  vec arr_host(MAX_ARR_SIZE);

  // Compute the size of the arrays in bytes for memory allocation.
  const size_t num_bytes = MAX_ARR_SIZE * sizeof(elem_type);

  // pointers to device arrays
  elem_type *device_input_array = nullptr;
  elem_type *device_output_array = nullptr;

  // TODO: allocate num_bytes of memory to the device arrays.
  // Hint: use hipMalloc

  // if either memory allocation failed, report an error message
  if (!device_input_array || !device_output_array) {
    cerr << "Couldn't allocate memory!" << endl;
    return 1;
  }

  // copy input to GPU
  hipMemcpy(device_input_array, &init_arr[0], num_bytes,
             hipMemcpyHostToDevice);
  check_launch("copy to gpu");

  /*
   * ––––––––––---------------------------
   * Questions 1.1 - 1.3: completing TODOs
   * ––––––––––---------------------------
   */

  // Testing accuracy of code

  size_t num_iter = 2;
  size_t array_size = 16;
  size_t cuda_block_size = 4;
  size_t cuda_grid_size = 4;
  host_recurrence(init_arr, arr_host, num_iter, array_size);
  recurAndCheck(device_input_array, device_output_array, num_iter, array_size,
                cuda_block_size, cuda_grid_size, arr_host);

  /* Further testing with more iterations */     
  array_size = 1e6;
  cuda_block_size = 1024;
  cuda_grid_size = 576;
  for (num_iter = 1; num_iter <= ITER_MAX_CHECK; ++num_iter) {
    host_recurrence(init_arr, arr_host, num_iter, array_size);
    recurAndCheck(device_input_array, device_output_array, num_iter, array_size,
                  cuda_block_size, cuda_grid_size, arr_host);
  }

  cout << "\nQuestions 1.1-1.3: your code passed all the tests!\n\n";

  // You can make the graph more easily by saving this array as a csv (or
  // something else)
  std::vector<double> performance_array;

  /*
   * ––––––––––-------------------------------------------------------
   * Question 1.4: vary number of threads for a small number of blocks
   * ––––––––––-------------------------------------------------------
   */
  cout << std::setw(23) << "Q1.4" << endl;
  cout << std::setw(43) << std::setfill('-') << " " << endl;
  cout << std::setw(15) << std::setfill(' ') << "Number of Threads";
  cout << std::setw(25) << "Performance TFlops/sec" << endl;
  cuda_grid_size = 72;
  num_iter = 4e4;
  array_size = 1e6;
  double flops = 2 * num_iter * array_size;
  host_recurrence(init_arr, arr_host, num_iter, array_size);
  for (size_t cuda_block_size = 32; cuda_block_size <= 1024;
       cuda_block_size += 32) {
    double elapsed_time =
        recurAndCheck(device_input_array, device_output_array, num_iter,
                      array_size, cuda_block_size, cuda_grid_size, arr_host);
    double performance = flops / (elapsed_time / 1000.) / 1E12;
    performance_array.push_back(performance);
    cout << std::setw(17) << cuda_block_size;
    cout << std::setw(25) << performance << endl;
    ;
  }
  cout << endl;
  performance_array.clear();

  /*
   * ––––––––––-------------------------------------------------------
   * Question 1.5: vary number of blocks for a small number of threads
   * ––––––––––-------------------------------------------------------
   */
  cout << std::setw(23) << "Q1.5" << endl;
  cout << std::setw(43) << std::setfill('-') << " " << endl;
  cout << std::setw(15) << std::setfill(' ') << "Number of Blocks";
  cout << std::setw(25) << "Performance TFlops/sec" << endl;
  cuda_block_size = 128;
  num_iter = 4e4;
  array_size = 1e6;
  flops = 2 * num_iter * array_size;
  host_recurrence(init_arr, arr_host, num_iter, array_size);
  for (size_t cuda_grid_size = 36; cuda_grid_size <= 1152;
       cuda_grid_size += 36) {
    double elapsed_time =
        recurAndCheck(device_input_array, device_output_array, num_iter,
                      array_size, cuda_block_size, cuda_grid_size, arr_host);
    double performance = flops / (elapsed_time / 1000.) / 1E12;
    performance_array.push_back(performance);
    cout << std::setw(16) << cuda_grid_size;
    cout << std::setw(25) << performance << endl;
    ;
  }
  cout << endl;
  performance_array.clear();

  /*
   * ––––––––––-----------------------------
   * Question 1.6: vary number of iterations
   * ––––––––––-----------------------------
   */
  cout << std::setw(23) << "Q1.6" << endl;
  cout << std::setw(43) << std::setfill('-') << " " << endl;
  cout << std::setw(15) << std::setfill(' ') << "Number of Iters";
  cout << std::setw(25) << "Performance TFlops/sec" << endl;
  cuda_block_size = 256;
  cuda_grid_size = 576;
  array_size = 1e6;
  std::vector<size_t> num_iters = {20,   40,   60,   80,   100,  120,  140,
                                   160,  180,  200,  300,  400,  500,  600,
                                   700,  800,  900,  1000, 1200, 1400, 1600,
                                   1800, 2000, 2200, 2400, 2600, 2800, 3000};
  for (size_t num_iter : num_iters) {
    flops = 2 * num_iter * array_size;
    host_recurrence(init_arr, arr_host, num_iter, array_size);
    double elapsed_time =
        recurAndCheck(device_input_array, device_output_array, num_iter,
                      array_size, cuda_block_size, cuda_grid_size, arr_host);
    double performance = flops / (elapsed_time / 1000.) / 1E12;
    performance_array.push_back(performance);
    cout << std::setw(15) << num_iter;
    cout << std::setw(25) << performance << endl;
  }
  cout << endl;
  performance_array.clear();

  // TODO: deallocate memory from both device arrays


  return exit_code;
}
