#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"

/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/

/*
  Caller function GEMM
*/
int myGEMM(nn_real* __restrict__ A, nn_real* __restrict__ B,
           nn_real* __restrict__ C, nn_real* alpha, nn_real* beta,
           int M, int N, int K) {

    // TODO
    return 0;
}


/* Helper functions for neural networks */
// TODO

